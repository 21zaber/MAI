#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <cassert>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <hiprand/hiprand_kernel.h>

#define CSC(call) {                                                                                                   \
    hipError_t err = call;                                                                                             \
    if (err != hipSuccess) {                                                                                         \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err));    \
        exit(1);                                                                                                      \
    }                                                                                                                 \
} while (0)

#define KERNEL_MIRROR(type, name) __constant__ type K_##name; type name
#define KERNEL_PUSH(val, type) CSC(hipMemcpyToSymbol(HIP_SYMBOL(K_ ## val), &val, sizeof(type))) 

//
// Configuration
//

// TODO
const int WIDTH =  1200;
const int HEIGHT = 700;
const int POINT_SIZE = 2;

// Formula's coefs
#define PSO_A1 0.1   // local coef
#define PSO_A2 0.01   // global coef
#define PSO_W  1.0
#define PSO_REP 1.0
KERNEL_MIRROR(float, PSO_DT) = 0.07;

#define INF 1000.0 * 1000.0 * 1000.0

// Function
#define Himmelblau(x, y) ((x*x + y - 11)*(x*x + y - 11) + (x + y*y - 7)*(x + y*y - 7))
#define Parabola(x, y) (x*x + y*y)
#define my_func(x, y) (sin((x+y)*0.001)*(x*x + y - 11)*(x*x + y - 11) + cos((x+y)*0.001)*(x + y*y - 7)*(x + y*y - 7))
//#define F(x, y) my_func(x, y)
#define F(x, y) Himmelblau(x, y)
//#define F(x, y) Parabola(x, y)

KERNEL_MIRROR(float, F_MAX);
KERNEL_MIRROR(float, F_MIN);
KERNEL_MIRROR(int2, F_X) = make_int2(-5, 5);
KERNEL_MIRROR(int2, F_Y) = make_int2(-5, 5);


// Current camera position
KERNEL_MIRROR(double, RADIUS) = 3;
KERNEL_MIRROR(double2, CENTER) = make_double2(0, 0);

#define RUN 1;
#define STOP 0;
int STATE = STOP;

const int P_NUM = 500;
__constant__ float2 *velocity;
__constant__ float2 *points;
__constant__ float3 *lbest;
__constant__ float3 gbest;
__constant__ float *fnc;

//
// Kernel calculations 
//

// P = plane, S = screen 
#define k_point_S2Px(xa) ((float)(xa-(WIDTH/2)) * ((float)K_RADIUS / WIDTH) * 2 + K_CENTER.x)
#define k_point_S2Py(ya) ((float)(ya-(HEIGHT/2)) * ((float)K_RADIUS / HEIGHT) * 2 + K_CENTER.y)

#define k_point_P2Sx(xa) (((float)xa - K_CENTER.x) * ((float)WIDTH / K_RADIUS) / 2 + WIDTH / 2)
#define k_point_P2Sy(ya) (((float)ya - K_CENTER.y) * ((float)HEIGHT / K_RADIUS) / 2 + HEIGHT / 2)

__device__ uchar4 k_get_color(float value) {
    const int color_num = 15;
    const uchar4 color[color_num] = {
        make_uchar4(0, 0, 0, 95), 
        make_uchar4(0, 0, 120, 95), 
        make_uchar4(50, 50, 255, 95), 
        make_uchar4(0, 150, 255, 95), 
        make_uchar4(0, 255, 255, 135), 
        make_uchar4(0, 255, 150, 135), 
        make_uchar4(0, 255, 50, 135), 
        make_uchar4(50, 255, 50, 175), 
        make_uchar4(150, 255, 0, 175), 
        make_uchar4(255, 255, 0, 215), 
        make_uchar4(255, 155, 0, 215), 
        make_uchar4(255, 55, 0, 215), 
        make_uchar4(255, 0, 0, 255),
        make_uchar4(255, 120, 120, 255),
        make_uchar4(255, 255, 255, 255)
    };
 
    int id1, id2;
    float coef = 0;
    if (value <= 0) {  
        id1 = id2 = 0;            
    } else if (value >= 1) {  
        id1 = id2 = color_num - 1; 
    } else {
        value = value * (color_num - 1);     
        id1 = (int)value;                
        id2 = id1 + 1;                      
        coef = value - id1;  
    }  
    uchar4 c1 = color[id1], c2 = color[id2], res = color[id1];
    res.x += coef * (c2.x - c1.x);
    res.y += coef * (c2.y - c1.y);
    res.z += coef * (c2.z - c1.z);
    res.w += coef * (c2.w - c1.w);
    return res;
}

__global__ void k_draw_map(uchar4* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    float x, y, f;
    for (int i = idx; i < WIDTH; i += offsetx) {
        x = k_point_S2Px(i);
        for (int j = idy; j < HEIGHT; j += offsety) {
            y = k_point_S2Py(j);
            f = F(x, y);
            float ratio = (f - K_F_MIN) / (K_F_MAX - K_F_MIN);
            data[j * WIDTH + i] = k_get_color(ratio);
        }
    }
}

__global__ void calc_fnc() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    float x, y;
    for (int i = idx; i < WIDTH; i += offsetx) {
        x = k_point_S2Px(i);
        for (int j = idy; j < HEIGHT; j += offsety) {
            y = k_point_S2Py(j);
            fnc[j * WIDTH + i] = F(x, y);
        }
    }
}

void calc_minmax() {
    // calc min and max function value on screen, using thrust
    void *tmp;
    hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(fnc), sizeof(float*));
    thrust::device_ptr<float> dev = thrust::device_pointer_cast((float*)tmp);
    thrust::pair<thrust::device_ptr<float>, thrust::device_ptr<float> > p = thrust::minmax_element(dev, dev + WIDTH * HEIGHT);
    F_MIN = p.first[0];
    F_MAX = p.second[0];
    KERNEL_PUSH(F_MIN, float);
    KERNEL_PUSH(F_MAX, float);
}

__global__ void draw_particle(uchar4* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    float x, y;
    int xr, yr;
    float r = POINT_SIZE*POINT_SIZE;
    uchar4 tmp;
    for (int p = idx; p < P_NUM; p += offsetx) {
        x = points[p].x;
        y = points[p].y;
        if (x <= K_CENTER.x - K_RADIUS || 
            x >= K_CENTER.x + K_RADIUS ||
            y <= K_CENTER.y - K_RADIUS || 
            y >= K_CENTER.y + K_RADIUS)  continue;
        xr = max(0.0, k_point_P2Sx(x));
        yr = max(0.0, k_point_P2Sy(y));
        for (int i = xr-POINT_SIZE; i <= xr+POINT_SIZE; i++) {
            for (int j = yr-POINT_SIZE; j <= yr+POINT_SIZE; j++) {
                if (((i-xr)*(i-xr) + (j-yr)*(j-yr) <= r) && i>0 && j>0 && i<WIDTH && j<HEIGHT) {
                    tmp = data[j * WIDTH + i];
                    data[j * WIDTH + i] = make_uchar4(255-tmp.x, 255-tmp.y, 255-tmp.z, 255);
                }
            }
        }
    }
}

__global__ void calc_lbest() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    float f;
    for (int i = idx; i < P_NUM; i += offsetx) {
        f = F(points[i].x, points[i].y);
        if (f < lbest[i].z) {
            lbest[i] = make_float3(points[i].x, points[i].y, f);
        }
    }
}

struct cmpr_lbest {
    __device__ bool operator()(float3 a, float3 b) {
        return a.z < b.z;
    }
};

void calc_gbest() {
    float3 *h_lbest;
    hipMemcpyFromSymbol(&h_lbest, HIP_SYMBOL(lbest), sizeof(float3*));
    thrust::device_ptr<float3> dev = thrust::device_pointer_cast(h_lbest);
    float3 extrm = thrust::min_element(dev, dev + P_NUM, cmpr_lbest())[0];

    float3 h_gbest;
    hipMemcpyFromSymbol(&h_gbest, HIP_SYMBOL(gbest), sizeof(float3));
    if (extrm.z < h_gbest.z) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(gbest), &extrm, sizeof(float3)));
        printf("Global min = %f in (%f, %f)\n", extrm.z, extrm.x, extrm.y);
    }
}

__global__ void calc_particle(unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    seed *= idx;
    hiprandState s;
    hiprand_init(seed, 0, 0, &s);

    float dt = K_PSO_DT;
    float a1 = PSO_A1 * dt, a2 = PSO_A2 * dt, w = PSO_W;
    float r1, r2;

    for (int p = idx; p < P_NUM; p += offsetx) {
        r1 = hiprand_uniform(&s);
        r2 = hiprand_uniform(&s);

        velocity[p].x = w * velocity[p].x +
            a1 * r1 * (lbest[p].x - points[p].x) +
            a2 * r2 * (gbest.x - points[p].x);

        r1 = hiprand_uniform(&s);
        r2 = hiprand_uniform(&s);

        velocity[p].y = w * velocity[p].y +
            a1 * r1 * (lbest[p].y - points[p].y) +
            a2 * r2 * (gbest.y - points[p].y);

        points[p].x += velocity[p].x * dt;
        points[p].y += velocity[p].y * dt;

        for (int i = 0; i < P_NUM; ++i){
            float2 p1 = points[p], p2 = points[i];
            if (i == p) continue;
            float r = (p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y);
            float2 v = make_float2(dt * (p1.x - p2.x) / (r * r), dt * (p1.y - p2.y) / (r * r));
            points[p].x += PSO_REP * v.x;
            points[p].y += PSO_REP * v.y;
        }
    }
}

__host__ __device__ float2 operator+(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

void calc_center() {
    void *h_points;
    hipMemcpyFromSymbol(&h_points, HIP_SYMBOL(points), sizeof(float2*));
    thrust::device_ptr<float2> dev = thrust::device_pointer_cast((float2*)h_points);
    float2 newCenter = thrust::reduce(dev, dev + P_NUM, make_float2(0.0, 0.0), thrust::plus<float2>());
    CENTER.x = newCenter.x / P_NUM;
    CENTER.y = newCenter.y / P_NUM;
    KERNEL_PUSH(CENTER, double2);
}

void random_particle() {
    srand(time(0));
    float2 *p = new float2[P_NUM];
    float2 *v = new float2[P_NUM];
    float3 *lb = new float3[P_NUM];
    void *tmp;

    for (int i = 0; i < P_NUM; i++) {
        p[i].x = ((float)rand() / (float)(RAND_MAX)) * (F_X.y - F_X.x) + F_X.x;
        p[i].y = ((float)rand() / (float)(RAND_MAX)) * (F_Y.y - F_Y.x) + F_Y.x;
        //v[i] = make_float2(0, 0);
        v[i] = make_float2(((float)rand() / (float)(RAND_MAX)), ((float)rand() / (float)(RAND_MAX)));
        lb[i] = make_float3(0, 0, INF);
    }

    CSC(hipMalloc(&tmp, sizeof(float2) * P_NUM));
    CSC(hipMemcpy(tmp, p, sizeof(float2) * P_NUM, hipMemcpyHostToDevice));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(points), &tmp, sizeof(float2*)));
    delete[] p;

    CSC(hipMalloc(&tmp, sizeof(float2) * P_NUM));
    CSC(hipMemcpy(tmp, v, sizeof(float2) * P_NUM, hipMemcpyHostToDevice));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(velocity), &tmp, sizeof(float2*)));
    delete[] v;

    CSC(hipMalloc(&tmp, sizeof(float3) * P_NUM));
    CSC(hipMemcpy(tmp, lb, sizeof(float3) * P_NUM, hipMemcpyHostToDevice));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(lbest), &tmp, sizeof(float3*)));
    delete[] lb;

    float3 h_gbest = make_float3(0.0, 0.0, INF);
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(gbest), &h_gbest, sizeof(float3)));

    CSC(hipMalloc(&tmp, sizeof(float) * WIDTH * HEIGHT));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(fnc), &tmp, sizeof(float*)));
}

//
// GL functions
//

struct hipGraphicsResource *res;  

void update() {
    uchar4* screen;
    size_t size;
    CSC(hipGraphicsMapResources(1, &res, 0));
    CSC(hipGraphicsResourceGetMappedPointer((void**)&screen, &size, res));

    calc_fnc<<<dim3(16, 16), dim3(16, 16)>>>();
    calc_minmax();
    k_draw_map<<<dim3(16, 16), dim3(16, 16)>>>(screen);
    draw_particle<<<16, 16>>>(screen);
  
    if (STATE) {
        calc_lbest<<<16, 16>>>();
        calc_gbest();
        calc_particle<<<16, 16>>>(rand());
        calc_center();
    }
  
    CSC(hipDeviceSynchronize());
    CSC(hipGraphicsUnmapResources(1, &res, 0));
  
    glutPostRedisplay();
}

void display() {
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glutSwapBuffers();
}

void processNormalKeys(unsigned char key, int x, int y) {
    switch (key) {
        case 'c':
            PSO_DT *= 0.7;
            break;
        case 'v':
            PSO_DT *= 1.4;
            break;
        case 'r':
            random_particle();
            break;
        case 'z':
            RADIUS = max(RADIUS * 0.7, 0.2);
            break;
        case 'x':
            RADIUS = RADIUS * 1.4;
            break;
        case ' ':
            STATE = !STATE;
            break;
        case 'd':
            CENTER.x += 0.5;
            break;
        case 'a':
            CENTER.x -= 0.5;
            break;
        case 'w':
            CENTER.y += 0.5;
            break;
        case 's':
            CENTER.y -= 0.5;
            break;
        case 'q':
            exit(0);
            break;
    }
    KERNEL_PUSH(PSO_DT, float);
    KERNEL_PUSH(RADIUS, double);
    KERNEL_PUSH(CENTER, double2);
}

//
// INIT functions
//

void init() {
    KERNEL_PUSH(PSO_DT, float);
    KERNEL_PUSH(F_X, int2);
    KERNEL_PUSH(F_Y, int2);
    KERNEL_PUSH(RADIUS, double);
    KERNEL_PUSH(CENTER, double2);
    random_particle();
}


int main(int argc, char** argv) {
    init();

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("CP N.I.Zabarin");

    glutIdleFunc(update);
    glutDisplayFunc(display);
    glutKeyboardFunc(processNormalKeys);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0.0, (GLdouble)WIDTH, 0.0, (GLdouble)HEIGHT);

    glewInit();

    GLuint vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIDTH * HEIGHT * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

    CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

    glutMainLoop();

    CSC(hipGraphicsUnregisterResource(res));

    glBindBuffer(1, vbo);
    glDeleteBuffers(1, &vbo);
}
