#include "hip/hip_runtime.h"
#include "conf.h"
//#include "gpu.h"

#include <stdio.h>
#include <cmath>
#include <cstdlib>
#include <ctime>

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

void display() {
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glutSwapBuffers();
}

int main(int argc, char** argv) {
    k_init();

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("CP N.I.Zabarin");

    glutIdleFunc(k_update);
    glutDisplayFunc(display);
    glutKeyboardFunc(k_processNormalKeys);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0.0, (GLdouble)WIDTH, 0.0, (GLdouble)HEIGHT);

    glewInit();

    GLuint vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIDTH * HEIGHT * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

    CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

    glutMainLoop();

    CSC(hipGraphicsUnregisterResource(res));

    glBindBuffer(1, vbo);
    glDeleteBuffers(1, &vbo);
}
