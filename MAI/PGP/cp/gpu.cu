#include "hip/hip_runtime.h"
#include "gpu.h"

//
// Kernel calculations 
//

__device__ uchar4 k_get_color(float value) {
    const int color_num = 15;
    const uchar4 color[color_num] = {
        make_uchar4(0, 0, 0, 95), 
        make_uchar4(0, 0, 120, 95), 
        make_uchar4(50, 50, 255, 95), 
        make_uchar4(0, 150, 255, 95), 
        make_uchar4(0, 255, 255, 135), 
        make_uchar4(0, 255, 150, 135), 
        make_uchar4(0, 255, 50, 135), 
        make_uchar4(50, 255, 50, 175), 
        make_uchar4(150, 255, 0, 175), 
        make_uchar4(255, 255, 0, 215), 
        make_uchar4(255, 155, 0, 215), 
        make_uchar4(255, 55, 0, 215), 
        make_uchar4(255, 0, 0, 255),
        make_uchar4(255, 120, 120, 255),
        make_uchar4(255, 255, 255, 255)
    };
 
    int id1, id2;
    float coef = 0;
    if (value <= 0) {  
        id1 = id2 = 0;            
    } else if (value >= 1) {  
        id1 = id2 = color_num - 1; 
    } else {
        value = value * (color_num - 1);     
        id1 = (int)value;                
        id2 = id1 + 1;                      
        coef = value - id1;  
    }  
    uchar4 c1 = color[id1], c2 = color[id2], res = color[id1];
    res.x += coef * (c2.x - c1.x);
    res.y += coef * (c2.y - c1.y);
    res.z += coef * (c2.z - c1.z);
    res.w += coef * (c2.w - c1.w);
    return res;
}

__global__ void k_draw_map(uchar4* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    float x, y, f;
    for (int i = idx; i < WIDTH; i += offsetx) {
        x = k_point_S2Px(i);
        for (int j = idy; j < HEIGHT; j += offsety) {
            y = k_point_S2Py(j);
            f = F(x, y);
            float ratio = (f - K_F_MIN) / (K_F_MAX - K_F_MIN);
            data[j * WIDTH + i] = k_get_color(ratio);
        }
    }
}

__global__ void k_calc_fnc() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    float x, y;
    for (int i = idx; i < WIDTH; i += offsetx) {
        x = k_point_S2Px(i);
        for (int j = idy; j < HEIGHT; j += offsety) {
            y = k_point_S2Py(j);
            fnc[j * WIDTH + i] = F(x, y);
        }
    }
}

void k_calc_minmax() {
    // calc min and max function value on screen, using thrust
    void *tmp;
    hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(fnc), sizeof(float*));
    thrust::device_ptr<float> dev = thrust::device_pointer_cast((float*)tmp);
    thrust::pair<thrust::device_ptr<float>, thrust::device_ptr<float> > p = thrust::minmax_element(dev, dev + WIDTH * HEIGHT);
    F_MIN = p.first[0];
    F_MAX = p.second[0];
    KERNEL_PUSH(F_MIN, float);
    KERNEL_PUSH(F_MAX, float);
}

__global__ void k_draw_particle(uchar4* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    float x, y;
    int xr, yr;
    float r = POINT_SIZE*POINT_SIZE;
    uchar4 tmp;
    for (int p = idx; p < P_NUM; p += offsetx) {
        x = points[p].x;
        y = points[p].y;
        if (x <= K_CENTER.x - K_RADIUS || 
            x >= K_CENTER.x + K_RADIUS ||
            y <= K_CENTER.y - K_RADIUS || 
            y >= K_CENTER.y + K_RADIUS)  continue;
        xr = max(0.0, k_point_P2Sx(x));
        yr = max(0.0, k_point_P2Sy(y));
        for (int i = xr-POINT_SIZE; i <= xr+POINT_SIZE; i++) {
            for (int j = yr-POINT_SIZE; j <= yr+POINT_SIZE; j++) {
                if (((i-xr)*(i-xr) + (j-yr)*(j-yr) <= r) && i>0 && j>0 && i<WIDTH && j<HEIGHT) {
                    tmp = data[j * WIDTH + i];
                    data[j * WIDTH + i] = make_uchar4(255-tmp.x, 255-tmp.y, 255-tmp.z, 255);
                }
            }
        }
    }
}

__global__ void k_calc_lbest() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    float f;
    for (int i = idx; i < P_NUM; i += offsetx) {
        f = F(points[i].x, points[i].y);
        if (f < lbest[i].z) {
            lbest[i] = make_float3(points[i].x, points[i].y, f);
        }
    }
}

struct cmpr_lbest {
    __device__ bool operator()(float3 a, float3 b) {
        return a.z < b.z;
    }
};

void k_calc_gbest() {
    float3 *h_lbest;
    hipMemcpyFromSymbol(&h_lbest, HIP_SYMBOL(lbest), sizeof(float3*));
    thrust::device_ptr<float3> dev = thrust::device_pointer_cast(h_lbest);
    float3 extrm = thrust::min_element(dev, dev + P_NUM, cmpr_lbest())[0];

    float3 h_gbest;
    hipMemcpyFromSymbol(&h_gbest, HIP_SYMBOL(gbest), sizeof(float3));
    if (extrm.z < h_gbest.z) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(gbest), &extrm, sizeof(float3)));
        printf("Global min = %f in (%f, %f)\n", extrm.z, extrm.x, extrm.y);
    }
}

__global__ void k_calc_particle(unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    seed *= idx;
    hiprandState s;
    hiprand_init(seed, 0, 0, &s);

    float dt = K_PSO_DT;
    float a1 = PSO_A1 * dt, a2 = PSO_A2 * dt, w = PSO_W;
    float r1, r2;

    for (int p = idx; p < P_NUM; p += offsetx) {
        r1 = hiprand_uniform(&s);
        r2 = hiprand_uniform(&s);

        velocity[p].x = w * velocity[p].x +
            a1 * r1 * (lbest[p].x - points[p].x) +
            a2 * r2 * (gbest.x - points[p].x);

        r1 = hiprand_uniform(&s);
        r2 = hiprand_uniform(&s);

        velocity[p].y = w * velocity[p].y +
            a1 * r1 * (lbest[p].y - points[p].y) +
            a2 * r2 * (gbest.y - points[p].y);

        points[p].x += velocity[p].x * dt;
        points[p].y += velocity[p].y * dt;

        for (int i = 0; i < P_NUM; ++i){
            float2 p1 = points[p], p2 = points[i];
            if (i == p) continue;
            float r = (p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y);
            float2 v = make_float2(dt * (p1.x - p2.x) / (r * r), dt * (p1.y - p2.y) / (r * r));
            points[p].x += PSO_REP * v.x;
            points[p].y += PSO_REP * v.y;
        }
    }
}

__host__ __device__ float2 operator+(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

void k_calc_center() {
    void *h_points;
    hipMemcpyFromSymbol(&h_points, HIP_SYMBOL(points), sizeof(float2*));
    thrust::device_ptr<float2> dev = thrust::device_pointer_cast((float2*)h_points);
    float2 newCenter = thrust::reduce(dev, dev + P_NUM, make_float2(0.0, 0.0), thrust::plus<float2>());
    CENTER.x = newCenter.x / P_NUM;
    CENTER.y = newCenter.y / P_NUM;
    KERNEL_PUSH(CENTER, double2);
}

void k_update() {
    uchar4* screen;
    size_t size;
    CSC(hipGraphicsMapResources(1, &res, 0));
    CSC(hipGraphicsResourceGetMappedPointer((void**)&screen, &size, res));

    k_calc_fnc<<<dim3(16, 16), dim3(16, 16)>>>();
    k_calc_minmax();
    k_draw_map<<<dim3(16, 16), dim3(16, 16)>>>(screen);
    k_draw_particle<<<16, 16>>>(screen);
  
    if (STATE) {
        k_calc_lbest<<<16, 16>>>();
        k_calc_gbest();
        k_calc_particle<<<16, 16>>>(rand());
        k_calc_center();
    }
  
    CSC(hipDeviceSynchronize());
    CSC(hipGraphicsUnmapResources(1, &res, 0));
  
    glutPostRedisplay();
}


void k_processNormalKeys(unsigned char key, int x, int y) {
    switch (key) {
        case 'c':
            PSO_DT *= 0.7;
            break;
        case 'v':
            PSO_DT *= 1.4;
            break;
        case 'r':
            k_random_particle();
            break;
        case 'z':
            RADIUS = max(RADIUS * 0.7, 0.2);
            break;
        case 'x':
            RADIUS = RADIUS * 1.4;
            break;
        case ' ':
            STATE = !STATE;
            break;
        case 'd':
            CENTER.x += 0.5;
            break;
        case 'a':
            CENTER.x -= 0.5;
            break;
        case 'w':
            CENTER.y += 0.5;
            break;
        case 's':
            CENTER.y -= 0.5;
            break;
        case 'q':
            exit(0);
            break;
    }
    KERNEL_PUSH(PSO_DT, float);
    KERNEL_PUSH(RADIUS, double);
    KERNEL_PUSH(CENTER, double2);
}
