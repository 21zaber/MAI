
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <iomanip>

using namespace std;

//
// DEFAULt functions for work with cuda
//

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)


double* dmalloc(int n) {
	double *d_arr;
	CSC(hipMalloc(&d_arr, sizeof(double) * n));
	CSC(hipGetLastError());
	return d_arr;
}

void dfree(double *darr) {
    CSC(hipFree(darr));
	CSC(hipGetLastError());
}

void hdmemcpy(double *arr, double *darr, int n) {
	CSC(hipMemcpy(darr, arr, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipGetLastError());
}

void dhmemcpy(double *arr, double *darr, int n) {
	CSC(hipMemcpy(arr, darr, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipGetLastError());
}


hipEvent_t start, stop;
float t;

void time_start() {
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));
}

void time_end() {
	CSC(hipGetLastError());
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&t, start, stop));	
	printf("time = %f\n", t);
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
}

//
// main programm
//


__global__ void kernel(double *darr, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;

	while (idx < n) {
		darr[idx] *= darr[idx];
		idx += offset;
	}
}

int main() {
	cout.precision(10);
	cout.setf(ios::scientific);

	int n;
	cin >> n;

	double *arr = new double[n];
	for (int i = 0; i < n; i++) {
		cin >> arr[i];
	}

	double *darr = dmalloc(n);
	hdmemcpy(arr, darr, n);

#ifdef TIMER
    timer_start();
#endif
    
	kernel<<<256, 256>>>(darr, n);

#ifdef TIMER
    timer_end();
#endif

	dhmemcpy(arr, darr, n);
	dfree(darr);

	for (int i = 0; i < n; i++) {
		cout << arr[i] << ' ';
	}
	cout << endl;
}
