
#include <hip/hip_runtime.h>
#include <cstdio>
#include <string>
#include <cassert>
#include <iostream>
#include <cstddef>
#include <vector>


#define uint8_t  unsigned char
#define uint16_t unsigned short
#define uint32_t unsigned int
#define uint64_t unsigned long long

using namespace std;
//
// DEFAULt functions for work with cuda
//

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)


hipEvent_t start, stop;
float t;

void time_start() {
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));
}

void time_end() {
	CSC(hipGetLastError());
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&t, start, stop));	
	printf("time = %f\n", t);
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
}

double minor3x3(double m[3][3], int i, int j) {
	double arr[4];
	int len = 0;
	for (int row = 0; row < 3; row++) {
		for (int col = 0; col < 3; col++) {
			if (col == j || row == i) continue;
			arr[len++] = m[row][col];
		}
	}
	return arr[0] * arr[3] - arr[1] * arr[2];
}

double cof3x3(double m[3][3], int i, int j) {
    double mr = minor3x3(m, i, j);
    if ((i+j) % 2 == 1)
        mr *= -1;
    return mr;
}

double det3x3(double m[3][3]) {
	double res = 0;
	for (int col = 0; col < 3; col++) {
		res += m[0][col] * cof3x3(m, 0, col);
	}
	return res;
}

class IMG {
public:
	uint32_t w;
    uint32_t h;
    uint64_t size;
	uchar4 *data;

	IMG() : w(0), h(0), size(0), data(NULL) {}

	IMG(string path) {
		FILE *fin = fopen(path.c_str(), "rb");
		if (!fin) {
            cout << "Wrong file name " << path << "!\n";
			return;
		}
		fread(&w, sizeof(uint32_t), 1, fin);
		fread(&h, sizeof(uint32_t), 1, fin);
        size = w * h;
		data = new uchar4[size];
		fread(data, sizeof(uchar4), size, fin);
		fclose(fin);
	}
	~IMG() {
		if (data != NULL) {
			delete[] data;
		}
	}

	void toFile(string path) {
		FILE *fout = fopen(path.c_str(), "wb");
		if (!fout) {
            cout << "Wrong file name " << path << "!\n";
			return;
		}
		fwrite(&w, sizeof(uint32_t), 1, fout);
		fwrite(&h, sizeof(uint32_t), 1, fout);
		fwrite(data, sizeof(uchar4), size, fout);
		fclose(fout);
	}
};

//
// main programm
//



#define pixel uchar4
#define p(x, y) data[y*w + x]

vector<vector<pixel> > cls;
__constant__ double GPU_AVG[50][3];
__constant__ double GPU_INVERT_COV[50][3][3];


__device__ double mahalanobis(const pixel p, int ci) {
	double v[3] = {p.x - GPU_AVG[ci][0], 
                   p.y - GPU_AVG[ci][1], 
                   p.z - GPU_AVG[ci][2] 
                  };
	double tmp[3] = {0, 0, 0}, res = 0;
	for (int i = 0; i < 3; ++i) {
		for (int j = 0; j < 3; ++j) {
			tmp[i] += v[j] * GPU_INVERT_COV[ci][j][i];
		}
        res -= tmp[i] * v[i];
	}
	return res;
}


__device__ void classify(pixel &p, int cnum) {
	double mx = mahalanobis(p, 0);
	p.w = 0;
	for (char ci = 1; ci < cnum; ++ci) {
		double tmp = mahalanobis(p, ci);
		if (tmp > mx) {
			mx = tmp;
			p.w = ci;
		}
	}
}

__global__ void kernel(uchar4 *data, uint32_t w, uint32_t h, int cnum) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for (int x = idx; x < w; x += offsetx) {
		for (int y = idy; y < h; y += offsety) {
			classify(p(x, y), cnum);
		}
	}
}

int main() {
int cnum = 0;
	string in, out;
	cin >> in >> out >> cnum;

    cls.resize(cnum, vector<pixel>());
    double cavg[cnum][3];
    double ccov[cnum][3][3];
    memset(cavg, 0, sizeof(double) * cnum * 3);
    memset(ccov, 0, sizeof(double) * cnum * 9);
	IMG img(in);
    uint32_t w = img.w; 

    for (int i = 0; i < cnum; ++i) {
        int csize, x, y;
        cin >> csize;
        for (int j = 0; j < csize; ++j) {
            cin >> x >> y;
            pixel p = img.p(x, y);
            cls[i].push_back(p);
            cavg[i][0] += p.x;    
            cavg[i][1] += p.y;    
            cavg[i][2] += p.z;    
        }
        for (int j = 0; j < 3; ++j) cavg[i][j] /= csize;
    }

	for (int c = 0; c < cnum; ++c) {
        int csize = cls[c].size();
		for (int i = 0; i < csize; ++i) {
			pixel p = cls[c][i];
			double coeff[3] = {p.x - cavg[c][0], 
                               p.y - cavg[c][1], 
                               p.z - cavg[c][2]
                              };

			for (int r = 0; r < 3; ++r) {
				for (int s = 0; s < 3; ++s) {
					ccov[c][r][s] += coeff[r] * coeff[s];
				}
			}
		}
        for (int r = 0; r < 3; ++r) {
            for (int s = 0; s < 3; ++s) {
				ccov[c][r][s] /= csize - 1;
			}
		}
		double det = det3x3(ccov[c]);
        double icov[3][3];
        for (int r = 0; r < 3; ++r) {
            for (int s = 0; s < 3; ++s) {
				if (det == 0) {
					icov[s][r] = (r == s ? 1 : 0);
				} else {
					icov[s][r] = cof3x3(ccov[c], r, s) / det;
				}
			}
		}
        for (int r = 0; r < 3; ++r) {
            for (int s = 0; s < 3; ++s) {
				ccov[c][r][s] = icov[r][s];
			}
		}
	}
	uchar4 *result;
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_AVG), cavg, sizeof(double) * cnum * 3));
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_INVERT_COV), ccov, sizeof(double) * cnum * 9));
	CSC(hipMalloc(&result, sizeof(uchar4) * img.size));
	CSC(hipMemcpy(result, img.data, sizeof(uchar4) * img.size, hipMemcpyHostToDevice));

	kernel<<<dim3(16, 16), dim3(16, 16)>>>(result, img.w, img.h, cnum);

	CSC(hipMemcpy(img.data, result, sizeof(uchar4) * img.size, hipMemcpyDeviceToHost));
	CSC(hipFree(result));

	img.toFile(out);
}
