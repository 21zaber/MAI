#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <cassert>
#include <iostream>
#include <cstddef>
#include <vector>                 
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <limits>

using namespace std;

#define int8_t  char
#define int16_t short
#define int32_t int
#define int64_t long long

//
// DEFAULt functions for work with cuda
//

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)


hipEvent_t start, stop;
float t;

void time_start() {
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));
}

void time_end() {
	CSC(hipGetLastError());
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&t, start, stop));	
	printf("time = %f\n", t);
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
}

const int BLOCK_SIZE = 1024;
const int GRID_SIZE = 16384;
//const int64_t INF = 2100000000;
const int64_t INF = 2147483647;

__global__ void k_blocksort(int64_t *arr, int len) {
    if (blockIdx.x * BLOCK_SIZE >= len) {
        return;
    }

    __shared__ int64_t block[BLOCK_SIZE];
    int idx = threadIdx.x * 2;
    int offset = blockIdx.x * BLOCK_SIZE;
    int nstep = BLOCK_SIZE / 2;

   // printf("offset %d", offset);

    block[idx] =  arr[offset + idx];
    block[idx+1] = arr[offset + idx+1];

    int64_t tmp;
 //         __syncthreads();
 // 		if (idx < 10)
 //         printf("%d, %d\n", idx, block[idx]);
    for (int k = 0; k < nstep; k++) {
        for (int i = idx+1; i < idx+3; ++i){
            __syncthreads();
            if (i < BLOCK_SIZE) {
                if (block[i-1] > block[i]) {
			//		printf("swap %d %d\n", i-1, i);
                    tmp = block[i-1];
                    block[i-1] = block[i];
                    block[i] = tmp;
                }
            }
        }
    }

    __syncthreads();
    arr[offset + idx] = block[idx];
    arr[offset + idx+1] = block[idx+1];
}


__global__ void k_merge(int64_t *arr, int len, bool odd) {
    int offset = blockIdx.x * BLOCK_SIZE * 2;
    int idx = threadIdx.x;
    int64_t tmp;

    if (odd) offset += BLOCK_SIZE; 

    if (offset + BLOCK_SIZE * 2 > len) return; 

    __shared__ int64_t block[BLOCK_SIZE * 2];

    int idt = 2 * idx;
    block[idt] =  arr[offset + idt];
    block[idt+1] = arr[offset + idt+1];

    __syncthreads();
    idt = BLOCK_SIZE * 2 - idx - 1;
    if (idx < BLOCK_SIZE && block[idx] > block[idt]) {
        tmp = block[idx];
        block[idx] = block[idt];
        block[idt] = tmp;
    }


    int step = BLOCK_SIZE / 2;
    while (step != 0) {
        __syncthreads();
        idt = idx;

        if ((idx / step) % 2 == 1) {
            idt -= step - BLOCK_SIZE;
        }
        if (block[idt] > block[idt + step]) {
            tmp = block[idt];
            block[idt] = block[idt+step];
            block[idt+step] = tmp;
        }
        step /= 2;
    }

    idt = 2 * idx;
    __syncthreads();
    arr[offset + idt] = block[idt];
    arr[offset + idt+1] = block[idt+1];
}


int main() {
    time_t tm, tmc;
    time(&tm);

    time(&tmc);
  //  fprintf(stderr, "%d: %f\t", __LINE__, difftime(tmc, tm));
    time(&tm);
    int n;
    fread(&n, sizeof(int), 1, stdin);

    fprintf(stderr, "n=%d\t", n);

    int rn = n;
    if (n % BLOCK_SIZE != 0) {
        n += BLOCK_SIZE - n % BLOCK_SIZE;
    }

    int64_t *arr = new int64_t[n];
    int l;
   // fprintf(stderr, "\n");
    for (int i = 0; i < n; i++) {
        l = INF;
        if (i < rn) {
            fread(&l, sizeof(int), 1, stdin);
            //fprintf(stderr, "%d ", l);
        }
        arr[i] = l;
    }
    //fprintf(stderr, "\n");
    

    time(&tmc);
  //  fprintf(stderr, "%d: %f\t", __LINE__, difftime(tmc, tm));
    time(&tm);

    int64_t *darr;
    CSC(hipMalloc(&darr, sizeof(int64_t) * n));
    CSC(hipMemcpy(darr, arr, sizeof(int64_t) * n, hipMemcpyHostToDevice));

    k_blocksort<<<GRID_SIZE, BLOCK_SIZE / 2>>>(darr, n);
    CSC(hipGetLastError());

  //CSC(hipMemcpy(arr, darr, sizeof(int64_t) * n, hipMemcpyDeviceToHost));
  //cout << "after sort" << endl;
  //for (int i = 0; i < rn; i++) {
  //    l = (int)arr[i];
  //    cout << l << " ";
  //    //fwrite(&l, sizeof(int), 1, stdout);
  //}
  //cout << endl;

    time(&tmc);
  //  fprintf(stderr, "%d: %f\t", __LINE__, difftime(tmc, tm));
    time(&tm);

	fprintf(stderr, "num of steps %d\n", n / BLOCK_SIZE);

    if (n > BLOCK_SIZE) {
        for (int step = 0; step < n / BLOCK_SIZE; step++) {
            k_merge<<<GRID_SIZE, BLOCK_SIZE>>>(darr, n, step & 1);
            CSC(hipGetLastError());
  //CSC(hipMemcpy(arr, darr, sizeof(int64_t) * n, hipMemcpyDeviceToHost));
  //cout << "after step" << endl;
  //for (int i = 0; i < rn; i++) {
  //    l = (int)arr[i];
  //    cout << l << " ";
  //    //fwrite(&l, sizeof(int), 1, stdout);
  //}
  //cout << endl;
        }
    }

    CSC(hipMemcpy(arr, darr, sizeof(int64_t) * rn, hipMemcpyDeviceToHost));
    CSC(hipFree(darr));

    time(&tmc);
  //  fprintf(stderr, "%d: %f\t", __LINE__, difftime(tmc, tm));
    time(&tm);

    for (int i = 0; i < rn; i++) {
        l = (int)arr[i];
      //  cout << l << " ";
        fwrite(&l, sizeof(int), 1, stdout);
    }
   // cout << endl;

    time(&tmc);
   // fprintf(stderr, "%d: %f\t", __LINE__, difftime(tmc, tm));
    time(&tm);
}
