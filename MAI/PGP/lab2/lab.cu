#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <cassert>
#include <iostream>
#include <cstddef>
//#include <cstdint>


#define uint8_t  unsigned char
#define uint16_t unsigned short
#define uint32_t unsigned int
#define uint64_t unsigned long long

using namespace std;
//
// DEFAULt functions for work with cuda
//

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

//
// main programm
//

uchar4 *readimg(string path, uint32_t &w, uint32_t &h) {
    FILE *f = fopen(path.c_str(), "rb");
    fread(&w, sizeof(uint32_t), 1, f);
    fread(&h, sizeof(uint32_t), 1, f);
    uint64_t size = w * h;     
    uchar4 *data = new uchar4[size];
    fread(data, sizeof(uchar4), size, f);
    fclose(f);
    return data;
}

void writeimg(string path, uchar4 *data, const uint32_t w, const uint32_t h) {
    FILE *fout = fopen(path.c_str(), "wb");
    fwrite(&w, sizeof(uint32_t), 1, fout);
    fwrite(&h, sizeof(uint32_t), 1, fout);
    uint64_t size = w * h;     
    fwrite(data, sizeof(uchar4), size, fout);
    fclose(fout);
    delete[] data;
}

__device__ uint8_t medc(uint16_t *cnt, int mid) {
	int sum = 0, i = -1;
	while (i < 256 && sum <= mid) sum += cnt[++i];
	return i;
}

texture<uchar4, 2, hipReadModeElementType> tex;

__device__ void medpx(uchar4 &px, int x0, int y0, uint32_t radius, uint32_t w, uint32_t h) {
	uchar4 p;
	uint16_t cnt[3][256];

    memset(cnt, 0, 3 * 256 * sizeof(uint16_t));

    int rd = (int)radius;

    int fx = max(0, x0-rd), tx = min(w, x0+rd+1);
    int fy = max(0, y0-rd), ty = min(h, y0+rd+1);
	int n = (tx - fx) * (ty - fy) / 2;

	for (int x = fx; x < tx; ++x) {
	    for (int y = fy; y < ty; ++y) {
			p = tex2D(tex, x, y);
			++cnt[0][p.x];
			++cnt[1][p.y];
			++cnt[2][p.z];
		}
	}


	px.x = medc(cnt[0], n);
	px.y = medc(cnt[1], n);
	px.z = medc(cnt[2], n);
	px.w = tex2D(tex, x0, y0).w;
}
                                        
__global__ void kernel(uchar4 *dst, uint32_t w, uint32_t h, uint32_t radius) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for (int x = idx; x < w; x += offsetx) {
		for (int y = idy; y < h; y += offsety) {
			 medpx(dst[y * w + x], x, y, radius, w, h);
		}
	}
}


int main() {
	string in, out;
	uint32_t radius, w, h;
	cin >> in >> out >> radius;

    uchar4 *data = readimg(in, w, h);
    uint64_t size = w * h;

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));
	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * size, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *result;
	CSC(hipMalloc(&result, sizeof(uchar4) * size));

	kernel<<<dim3(16, 16), dim3(16, 16)>>>(result, w, h, radius);
	CSC(hipGetLastError());
	CSC(hipMemcpy(data, result, sizeof(uchar4) * size, hipMemcpyDeviceToHost));

	CSC(hipUnbindTexture(tex));
	CSC(hipFreeArray(arr));
	CSC(hipFree(result));

	writeimg(out, data, w, h);
}
